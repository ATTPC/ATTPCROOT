#include "hip/hip_runtime.h"
#ifndef _TRACKFIT_KERNEL_H_
#define _TRACKFIT_KERNEL_H_

#include <stdio.h>  

#include <hip/hip_runtime.h>  
#include <cutil.h>

inline __device__ int Newton(double Mxx, double Myy, double Mxy, double Mxz, double Myz, double Mzz,double *result)

{
 
  double Mz,Mxz2,Myz2,Cov_xy; //,temp;
  double A0, A1, A2, A22, epsilon = 0.000000000001;
  double Dy, xnew, xold, ynew, yold = 100000000000.;



  Mz = Mxx + Myy;
  Cov_xy = Mxx*Myy - Mxy*Mxy;
  Mxz2 = Mxz*Mxz;
  Myz2 = Myz*Myz;

  A2 = 4.*Cov_xy - 3.*Mz*Mz - Mzz;
  A1 = Mzz*Mz + 4.*Cov_xy*Mz - Mxz2 - Myz2 - Mz*Mz*Mz;
  A0 = Mxz2*Myy + Myz2*Mxx - Mzz*Cov_xy - 2.*Mxz*Myz*Mxy + Mz*Mz*Cov_xy;

  A22 = A2 + A2;
  //  iter = 0;
  xnew = 0.;

  //    Newton's method starting at x=0

  int iter, iterMax = 20;
  for (iter=0; iter < iterMax; iter++) {
    ynew = A0 + xnew*(A1 + xnew*(A2 + 4.*xnew*xnew));

    if (fabs(ynew)>fabs(yold)) {
      //          printf("Newton2 goes wrong direction: ynew=%f
      //          yold=%f\n",ynew,yold);
      xnew = 0.;
      break;
    }

    Dy = A1 + xnew*(A22 + 16.*xnew*xnew);
    xold = xnew;
    xnew = xold - ynew/Dy;
    
    if (fabs(xnew) < epsilon) break;  
    if (fabs((xnew-xold)/xnew) < epsilon) break;
  }

  if (iter == iterMax-1) {
    //      printf("Newton2 does not converge in %d
    //      iterations\n",iterMax);
    xnew = 0.;
  }

  if (xnew < 0.) {
    iter=30;
         // printf("Negative root: x=%f\n",xnew);
  }

  //    computing the circle parameters

  double GAM,DET;
 // double Xcenter,Ycenter,Radius;

  GAM = - Mz - xnew - xnew;
  DET = xnew*xnew - xnew*Mz + Cov_xy;
  if (DET==0) return 0;

  result[0] = (Mxz*(Myy-xnew) - Myz*Mxy)/DET/2.;
  result[1] = (Myz*(Mxx-xnew) - Mxz*Mxy)/DET/2.;
  if ((result[0]*result[0]+result[1]*result[1]-GAM) < 0.) return 0;

  result[2] = sqrt(result[0]*result[0]+result[1]*result[1]-GAM);
   
 
   return 1;
 
}



__global__ void 
 
Fit(double *X, double *Y,double *Z1, double *Z1err, double *Mx,double *My,double *M0,double *result)
{
   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
  // unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

   // printf("Negative root: x=%f\n");

   __shared__ double Xs[30];
   __shared__ double Ys[30];
   __shared__ double Xis[30];
   __shared__ double Yis[30];
   __shared__ double Zis[30];
   __shared__ double Z1s[30];
   __shared__ double Z1Errs[30];
   __shared__ double rho[30];
   __shared__ double fZWeight[30]; 

   if(x < 30){
           Xs[x]    = X[x];
           Ys[x]    = Y[x];
           Z1s[x]   = Z1[x];
           Z1Errs[x]= Z1err[x];
        //   printf("Z1Errs(%d) = %g \n",x , Z1Errs[x]);
   }

   Xis[x] = Xs[x]- Mx[0];
   Yis[x] = Ys[x]- My[0];
   
   __syncthreads();
   
   Zis[x] = Xis[x]*Xis[x] + Yis[x]*Yis[x];
   rho[x] = sqrt(Xis[x]*Xis[x] + Yis[x]*Yis[x]);
 //  printf("rho(%d) = %g \n",x , rho[x]); 
   if(Z1Errs[x]>0.001)fZWeight[x] = 1/(Z1Errs[x]*Z1Errs[x]);
   else {
   //  printf("Z1Err(%d) = %g \n",x , Z1Errs[x]); 
     fZWeight[x] = 0.0;
   
   }
  
   __syncthreads();

  if(x<1) { 
   double Mxx=0;
   double Myy=0;
   double Mxy=0;
   double Mxz=0;
   double Myz=0;
   double Mzz=0;



   double wsum  = 0. ;
   double wx  = 0. ;
   double wy  = 0. ;
   double wxx = 0.;
   double wxy = 0. ;

   for(int i=0; i<30; i++){
     Mxy += Xis[i]*Yis[i];
     Mxx += Xis[i]*Xis[i];
     Myy += Yis[i]*Yis[i];
     Mxz += Xis[i]*Zis[i];
     Myz += Yis[i]*Zis[i];
     Mzz += Zis[i]*Zis[i];
     
     wsum+= fZWeight[i];
     wx  += fZWeight[i]*rho[i];
     wy  += fZWeight[i]*Z1s[i];
     wxx += fZWeight[i]*rho[i]*rho[i];
     wxy += fZWeight[i]*rho[i]*Z1s[i];
  //   printf("Weights    =%f   %f  %f  \n" ,fZWeight[i],rho[i],Z1s[i] );
   }
    
   Mxx /= M0[0];
   Myy /= M0[0];
   Mxy /= M0[0];
   Mxz /= M0[0];
   Myz /= M0[0];
   Mzz /= M0[0];
   
//  printf("Result =%f   %f  %f %f  %f  \n" ,wsum, wxy, wxx, wx, wy );
   int t= Newton(Mxx, Myy, Mxy, Mxz, Myz, Mzz, result);
   

  double mm = 0.;
  double qq = 0.;  
  double det = wsum * wxx - wx * wx;
   if (det>0.00001) {
      mm = (wxy * wsum - wy * wx) / det;
      qq = (wy * wxx - wxy * wx) / det;
   }else {mm=1000.; qq=1000.;} 

   result[3]=-mm;
   result[4]=qq;
 

   double r1 ;
   double chi2 = 0.;
  
   for(int i=0; i < 30; i++) {
     r1   = Z1s[i] + result[3] * rho[i] - result[4];
     chi2 += fZWeight[i] * (r1 * r1);
   }
   result[5]=chi2;


   if (det>0.00001){
     double varsq = sqrt(chi2/30.);
     result[6] = varsq * sqrt( wsum / det );
     result[7] = varsq * sqrt( wxx / det );
   }else{
     result[6] = 0;
     result[7] = 0;
   }

  }
  
  
}



//--------------------------------------------------------------------------------------------------------------//



inline __device__ int NewtonF(float Mxx, float Myy, float Mxy, float Mxz, float Myz, float Mzz,float *result)

{
 
  float Mz,Mxz2,Myz2,Cov_xy; //,temp;
  float A0, A1, A2, A22, epsilon = 0.000000000001;
  float Dy, xnew, xold, ynew, yold = 100000000000.;



  Mz = Mxx + Myy;
  Cov_xy = Mxx*Myy - Mxy*Mxy;
  Mxz2 = Mxz*Mxz;
  Myz2 = Myz*Myz;

  A2 = 4.*Cov_xy - 3.*Mz*Mz - Mzz;
  A1 = Mzz*Mz + 4.*Cov_xy*Mz - Mxz2 - Myz2 - Mz*Mz*Mz;
  A0 = Mxz2*Myy + Myz2*Mxx - Mzz*Cov_xy - 2.*Mxz*Myz*Mxy + Mz*Mz*Cov_xy;

  A22 = A2 + A2;
  //  iter = 0;
  xnew = 0.;

  //    Newton's method starting at x=0

  int iter, iterMax = 20;
  for (iter=0; iter < iterMax; iter++) {
    ynew = A0 + xnew*(A1 + xnew*(A2 + 4.*xnew*xnew));

    if (fabs(ynew)>fabs(yold)) {
      //          printf("Newton2 goes wrong direction: ynew=%f
      //          yold=%f\n",ynew,yold);
      xnew = 0.;
      break;
    }

    Dy = A1 + xnew*(A22 + 16.*xnew*xnew);
    xold = xnew;
    xnew = xold - ynew/Dy;
    
    if (fabs(xnew) < epsilon) break;  
    if (fabs((xnew-xold)/xnew) < epsilon) break;
  }

  if (iter == iterMax-1) {
    //      printf("Newton2 does not converge in %d
    //      iterations\n",iterMax);
    xnew = 0.;
  }

  if (xnew < 0.) {
    iter=30;
         // printf("Negative root: x=%f\n",xnew);
  }

  //    computing the circle parameters

  float GAM,DET;
 // float Xcenter,Ycenter,Radius;

  GAM = - Mz - xnew - xnew;
  DET = xnew*xnew - xnew*Mz + Cov_xy;
  if (DET==0) return 0;

  result[0] = (Mxz*(Myy-xnew) - Myz*Mxy)/DET/2.;
  result[1] = (Myz*(Mxx-xnew) - Mxz*Mxy)/DET/2.;
  if ((result[0]*result[0]+result[1]*result[1]-GAM) < 0.) return 0;

  result[2] = sqrt(result[0]*result[0]+result[1]*result[1]-GAM);
   
 
   return 1;
 
}



__global__ void 
 
FitF(float *X, float *Y,float *Z1, float *Z1err, float *Mx1,float *My1,float *M01,float *result)
{
   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
  // unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

   // printf("Negative root: x=%f\n");

   __shared__ float Xs[15];
   __shared__ float Ys[15];
   __shared__ float Xis[15];
   __shared__ float Yis[15];
   __shared__ float Zis[15];
   __shared__ float Z1s[15];
   __shared__ float Z1Errs[15];
   __shared__ float rho[15];
   __shared__ float fZWeight[15]; 
   __shared__ float Mx[1]; 
   __shared__ float My[1]; 
   __shared__ float M0[1]; 

//   if(x < 30){
           Xs[x]    = X[x];
           Ys[x]    = Y[x];
           Z1s[x]   = Z1[x];
           Z1Errs[x]= Z1err[x];
        //   printf("Z1Errs(%d) = %g \n",x , Z1Errs[x]);
//   }
   if(x<1){
      Mx[0]=Mx1[0]; 
      My[0]=My1[0]; 
      M0[0]=M01[0]; 
   }
   __syncthreads();
   Xis[x] = Xs[x]- Mx[0];
   Yis[x] = Ys[x]- My[0];
   Zis[x] = Xis[x]*Xis[x] + Yis[x]*Yis[x];
   rho[x] = sqrt(Xis[x]*Xis[x] + Yis[x]*Yis[x]);
 //  printf("rho(%d) = %g \n",x , rho[x]); 
   if(Z1Errs[x]>0.001)fZWeight[x] = 1/(Z1Errs[x]*Z1Errs[x]);
   else {
   //  printf("Z1Err(%d) = %g \n",x , Z1Errs[x]); 
     fZWeight[x] = 0.0;
   }

   __syncthreads();

  if(x<1) { 
   float Mxx=0;
   float Myy=0;
   float Mxy=0;
   float Mxz=0;
   float Myz=0;
   float Mzz=0;



   float wsum  = 0. ;
   float wx  = 0. ;
   float wy  = 0. ;
   float wxx = 0.;
   float wxy = 0. ;

   for(int i=0; i<15; i++){
     Mxy += Xis[i]*Yis[i];
     Mxx += Xis[i]*Xis[i];
     Myy += Yis[i]*Yis[i];
     Mxz += Xis[i]*Zis[i];
     Myz += Yis[i]*Zis[i];
     Mzz += Zis[i]*Zis[i];
     
     wsum+= fZWeight[i];
     wx  += fZWeight[i]*rho[i];
     wy  += fZWeight[i]*Z1s[i];
     wxx += fZWeight[i]*rho[i]*rho[i];
     wxy += fZWeight[i]*rho[i]*Z1s[i];
  //   printf("Weights    =%f   %f  %f  \n" ,fZWeight[i],rho[i],Z1s[i] );
   }
    
   Mxx /= M0[0];
   Myy /= M0[0];
   Mxy /= M0[0];
   Mxz /= M0[0];
   Myz /= M0[0];
   Mzz /= M0[0];
   
//  printf("Result =%f   %f  %f %f  %f  \n" ,wsum, wxy, wxx, wx, wy );
   int t= NewtonF(Mxx, Myy, Mxy, Mxz, Myz, Mzz, result);
   

  float mm = 0.;
  float qq = 0.;  
  float det = wsum * wxx - wx * wx;
   if (det>0.00001) {
      mm = (wxy * wsum - wy * wx) / det;
      qq = (wy * wxx - wxy * wx) / det;
   }else {mm=1000.; qq=1000.;} 

   result[3]=-mm;
   result[4]=qq;
 

   float r1 ;
   float chi2 = 0.;
  
   for(int i=0; i < 15; i++) {
     r1   = Z1s[i] + result[3] * rho[i] - result[4];
     chi2 += fZWeight[i] * (r1 * r1);
   }
   result[5]=chi2;


   if (det>0.00001){
     float varsq = sqrt(chi2/13.);
     result[6] = varsq * sqrt( wsum / det );
     result[7] = varsq * sqrt( wxx / det );
   }else{
     result[6] = 0;
     result[7] = 0;
   }

  }
  
  
}





inline __device__ int NewtonAllF(float Mxx, float Myy, float Mxy, float Mxz, float Myz, float Mzz,float *result)

{
  unsigned int y = blockIdx.x;
  float Mz,Mxz2,Myz2,Cov_xy; //,temp;
  float A0, A1, A2, A22, epsilon = 0.000000000001;
  float Dy, xnew, xold, ynew, yold = 100000000000.;



  Mz = Mxx + Myy;
  Cov_xy = Mxx*Myy - Mxy*Mxy;
  Mxz2 = Mxz*Mxz;
  Myz2 = Myz*Myz;

  A2 = 4.*Cov_xy - 3.*Mz*Mz - Mzz;
  A1 = Mzz*Mz + 4.*Cov_xy*Mz - Mxz2 - Myz2 - Mz*Mz*Mz;
  A0 = Mxz2*Myy + Myz2*Mxx - Mzz*Cov_xy - 2.*Mxz*Myz*Mxy + Mz*Mz*Cov_xy;

  A22 = A2 + A2;
  //  iter = 0;
  xnew = 0.;

  //    Newton's method starting at x=0

  int iter, iterMax = 20;
  for (iter=0; iter < iterMax; iter++) {
    ynew = A0 + xnew*(A1 + xnew*(A2 + 4.*xnew*xnew));

    if (fabs(ynew)>fabs(yold)) {
      //          printf("Newton2 goes wrong direction: ynew=%f
      //          yold=%f\n",ynew,yold);
      xnew = 0.;
      break;
    }

    Dy = A1 + xnew*(A22 + 16.*xnew*xnew);
    xold = xnew;
    xnew = xold - ynew/Dy;
    
    if (fabs(xnew) < epsilon) break;  
    if (fabs((xnew-xold)/xnew) < epsilon) break;
  }

  if (iter == iterMax-1) {
    //      printf("Newton2 does not converge in %d
    //      iterations\n",iterMax);
    xnew = 0.;
  }

  if (xnew < 0.) {
    iter=30;
         // printf("Negative root: x=%f\n",xnew);
  }

  //    computing the circle parameters

  float GAM,DET;
 // float Xcenter,Ycenter,Radius;

  GAM = - Mz - xnew - xnew;
  DET = xnew*xnew - xnew*Mz + Cov_xy;
  if (DET==0) return 0;

  result[8*y] = (Mxz*(Myy-xnew) - Myz*Mxy)/DET/2.;
  result[1+8*y] = (Myz*(Mxx-xnew) - Mxz*Mxy)/DET/2.;
  if ((result[8*y]*result[8*y]+result[1+8*y]*result[1+8*y]-GAM) < 0.) return 0;

  result[2+8*y] = sqrt(result[8*y]*result[8*y]+result[1+8*y]*result[1+8*y]-GAM);
   
 
   return 1;
 
}









__global__ void 
FitAllF(float *X, float *Y,float *Z1, float *Z1err, float *Mx1,float *My1,float *M01,float *result)
{
//   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int y = blockIdx.x;
   unsigned int x = threadIdx.x;
  // unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
 //   printf("Block No: =%d , threadId.x =%d \n",y,x);
   __shared__ float Xs[30];
   __shared__ float Ys[30];
   __shared__ float Xis[30];
   __shared__ float Yis[30];
   __shared__ float Zis[30];
   __shared__ float Z1s[30];
   __shared__ float Z1Errs[30];
   __shared__ float rho[30];
   __shared__ float fZWeight[30];   
   __shared__ float Mx[100]; 
   __shared__ float My[100]; 
   __shared__ float M0[100]; 

   float Mxx=0.;
   float Myy=0.;
   float Mxy=0.;
   float Mxz=0.;
   float Myz=0.;
   float Mzz=0.;
   float wsum= 0. ;
   float wx  = 0. ;
   float wy  = 0. ;
   float wxx = 0.;
   float wxy = 0. ;


   Xs[x]    = X[x+30*y];
   Ys[x]    = Y[x+30*y];
   Z1s[x]   = Z1[x+30*y];
   Z1Errs[x]= Z1err[x+30*y];
  //         printf("blockid = %d  Z1Errs(%d) = %g \n",y, x , Z1Errs[x]);
  //   if(x < 8) result[x+8*y]=0.000;
   
   if(x<1){
      Mx[y]=Mx1[y]; 
      My[y]=My1[y]; 
      M0[y]=M01[y]; 
   }
   __syncthreads();

   Xis[x] = Xs[x]- Mx[y];
   Yis[x] = Ys[x]- My[y];
   Zis[x] = Xis[x]*Xis[x] + Yis[x]*Yis[x];
   rho[x] = sqrt(Zis[x]);

   Mxy+= Xis[x]*Yis[x]/M0[y];
   Mxx+= Xis[x]*Xis[x]/M0[y];
   Myy+= Yis[x]*Yis[x]/M0[y];
   Mxz+= Xis[x]*Zis[x]/M0[y];
   Myz+= Yis[x]*Zis[x]/M0[y];
   Mzz+= Zis[x]*Zis[x]/M0[y];

 
 //  printf("rho(%d) = %g \n",x , rho[x]); 
   if(Z1Errs[x]>0.001)fZWeight[x] = 1/(Z1Errs[x]*Z1Errs[x]);
   else {
   //  printf("Z1Err(%d) = %g \n",x , Z1Errs[x]); 
     fZWeight[x] = 0.0;
   
   }
 
   wsum+= fZWeight[x];
   wx  += fZWeight[x]*rho[x];
   wy  += fZWeight[x]*Z1s[x];
   wxx += fZWeight[x]*Z1s[x];
   wxy += fZWeight[x]*rho[x]*Z1s[x];

 
   __syncthreads();

  if(x<1) { 
//  printf("Result =%f   %f  %f %f  %f  \n" ,wsum, wxy, wxx, wx, wy );
   int t= NewtonAllF(Mxx, Myy, Mxy, Mxz, Myz, Mzz, result);
  float mm = 0.;
  float qq = 0.;  
  float det = wsum * wxx - wx * wx;
   if (det>0.00001) {
      mm = (wxy * wsum - wy * wx) / det;
      qq = (wy * wxx - wxy * wx) / det;
   }else {mm=1000.; qq=1000.;} 

   result[3+8*y]=-mm;
   result[4+8*y]=qq;
 
   float r1 ;
   float chi2 = 0.;
  
   for(int i=0; i < 30; i++) {
     r1   = Z1s[i] + result[3+8*y] * rho[i] - result[4+8*y];
     chi2 += fZWeight[i] * (r1 * r1);
   }
   result[5+8*y]=chi2;

   if (det>0.00001){
     float varsq = sqrt(chi2/25.);
     result[6+8*y] = varsq * sqrt( wsum / det );
     result[7+8*y] = varsq * sqrt( wxx / det );
   }else{
     result[6+8*y] = 0;
     result[7+8*y] = 0;
   }

  }
  
  
}



inline __device__ int NewtonAllD(double Mxx, double Myy, double Mxy, double Mxz, double Myz, double Mzz,double *result)

{
  unsigned int y = blockIdx.x;
  double Mz,Mxz2,Myz2,Cov_xy; //,temp;
  double A0, A1, A2, A22, epsilon = 0.000000000001;
  double Dy, xnew, xold, ynew, yold = 100000000000.;



  Mz = Mxx + Myy;
  Cov_xy = Mxx*Myy - Mxy*Mxy;
  Mxz2 = Mxz*Mxz;
  Myz2 = Myz*Myz;

  A2 = 4.*Cov_xy - 3.*Mz*Mz - Mzz;
  A1 = Mzz*Mz + 4.*Cov_xy*Mz - Mxz2 - Myz2 - Mz*Mz*Mz;
  A0 = Mxz2*Myy + Myz2*Mxx - Mzz*Cov_xy - 2.*Mxz*Myz*Mxy + Mz*Mz*Cov_xy;

  A22 = A2 + A2;
  //  iter = 0;
  xnew = 0.;

  //    Newton's method starting at x=0

  int iter, iterMax = 20;
  for (iter=0; iter < iterMax; iter++) {
    ynew = A0 + xnew*(A1 + xnew*(A2 + 4.*xnew*xnew));

    if (fabs(ynew)>fabs(yold)) {
      //          printf("Newton2 goes wrong direction: ynew=%f
      //          yold=%f\n",ynew,yold);
      xnew = 0.;
      break;
    }

    Dy = A1 + xnew*(A22 + 16.*xnew*xnew);
    xold = xnew;
    xnew = xold - ynew/Dy;
    
    if (fabs(xnew) < epsilon) break;  
    if (fabs((xnew-xold)/xnew) < epsilon) break;
  }

  if (iter == iterMax-1) {
    //      printf("Newton2 does not converge in %d
    //      iterations\n",iterMax);
    xnew = 0.;
  }

  if (xnew < 0.) {
    iter=30;
         // printf("Negative root: x=%f\n",xnew);
  }

  //    computing the circle parameters

  double GAM,DET;
 // double Xcenter,Ycenter,Radius;

  GAM = - Mz - xnew - xnew;
  DET = xnew*xnew - xnew*Mz + Cov_xy;
  if (DET==0) return 0;

  result[8*y] = (Mxz*(Myy-xnew) - Myz*Mxy)/DET/2.;
  result[1+8*y] = (Myz*(Mxx-xnew) - Mxz*Mxy)/DET/2.;
  if ((result[8*y]*result[8*y]+result[1+8*y]*result[1+8*y]-GAM) < 0.) return 0;

  result[2+8*y] = sqrt(result[8*y]*result[8*y]+result[1+8*y]*result[1+8*y]-GAM);
   
 
   return 1;
 
}









__global__ void 
FitAllD(double *X, double *Y,double *Z1, double *Z1err, double *Mx,double *My,double *M0,double *result)
{
//   unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int y = blockIdx.x;
   unsigned int x = threadIdx.x;
  // unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
 //   printf("Block No: =%d , threadId.x =%d \n",y,x);
   __shared__ double Xs[30];
   __shared__ double Ys[30];
   __shared__ double Xis[30];
   __shared__ double Yis[30];
   __shared__ double Zis[30];
   __shared__ double Z1s[30];
   __shared__ double Z1Errs[30];
   __shared__ double rho[30];
   __shared__ double fZWeight[30];   
 
   double Mxx=0.;
   double Myy=0.;
   double Mxy=0.;
   double Mxz=0.;
   double Myz=0.;
   double Mzz=0.;
   double wsum= 0. ;
   double wx  = 0. ;
   double wy  = 0. ;
   double wxx = 0.;
   double wxy = 0. ;


   Xs[x]    = X[x+30*y];
   Ys[x]    = Y[x+30*y];
   Z1s[x]   = Z1[x+30*y];
   Z1Errs[x]= Z1err[x+30*y];
  //         printf("blockid = %d  Z1Errs(%d) = %g \n",y, x , Z1Errs[x]);
  //   if(x < 8) result[x+8*y]=0.000;
   
  
   __syncthreads();

   Xis[x] = Xs[x]- Mx[y];
   Yis[x] = Ys[x]- My[y];
   Zis[x] = Xis[x]*Xis[x] + Yis[x]*Yis[x];
   rho[x] = sqrt(Zis[x]);

   Mxy+= Xis[x]*Yis[x]/M0[y];
   Mxx+= Xis[x]*Xis[x]/M0[y];
   Myy+= Yis[x]*Yis[x]/M0[y];
   Mxz+= Xis[x]*Zis[x]/M0[y];
   Myz+= Yis[x]*Zis[x]/M0[y];
   Mzz+= Zis[x]*Zis[x]/M0[y];

 
 //  printf("rho(%d) = %g \n",x , rho[x]); 
   if(Z1Errs[x]>0.001)fZWeight[x] = 1/(Z1Errs[x]*Z1Errs[x]);
   else {
   //  printf("Z1Err(%d) = %g \n",x , Z1Errs[x]); 
     fZWeight[x] = 0.0;
   
   }
 
   wsum+= fZWeight[x];
   wx  += fZWeight[x]*rho[x];
   wy  += fZWeight[x]*Z1s[x];
   wxx += fZWeight[x]*Z1s[x];
   wxy += fZWeight[x]*rho[x]*Z1s[x];

 
   __syncthreads();

  if(x<1) { 
//  printf("Result =%f   %f  %f %f  %f  \n" ,wsum, wxy, wxx, wx, wy );
   int t= NewtonAllD(Mxx, Myy, Mxy, Mxz, Myz, Mzz, result);
  double mm = 0.;
  double qq = 0.;  
  double det = wsum * wxx - wx * wx;
   if (det>0.00001) {
      mm = (wxy * wsum - wy * wx) / det;
      qq = (wy * wxx - wxy * wx) / det;
   }else {mm=1000.; qq=1000.;} 

   result[3+8*y]=-mm;
   result[4+8*y]=qq;
 
   double r1 ;
   double chi2 = 0.;
  
   for(int i=0; i < 30; i++) {
     r1   = Z1s[i] + result[3+8*y] * rho[i] - result[4+8*y];
     chi2 += fZWeight[i] * (r1 * r1);
   }
   result[5+8*y]=chi2;

   if (det>0.00001){
     double varsq = sqrt(chi2/25.);
     result[6+8*y] = varsq * sqrt( wsum / det );
     result[7+8*y] = varsq * sqrt( wxx / det );
   }else{
     result[6+8*y] = 0;
     result[7+8*y] = 0;
   }

  }
  
  
}




#endif // #ifndef _TRACKFIT_KERNEL_H_ 










