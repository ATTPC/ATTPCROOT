
#include <stdio.h>  
#include <assert.h>  
#include <hip/hip_runtime.h>  
#include <cutil.h>
#include <trackfit_kernel.cu>
#include "HitTrk.h"



extern "C" void CircleFitG(double X[HIT], double Y[HIT], double Z[HIT], double Zerr[HIT], double *Mx,double *My,double *M0,double *result)
{
//   printf(" Now in Cuda :  Mx =  %g  :  My =  %g  \n", Mx[0],My[0] );
 
 /*  for(int i=0; i<50; i++){
       printf(" Zerr[%i] =  %g  :  Z[%i] =  %g  \n", i, Zerr[i],i ,Z[i]  );
   }
*/
   double *d_X;
   double *d_Y;
   double *d_Z;
   double *d_Zerr;
   double *d_Mx;
   double *d_My;
   double *d_M0;
   double *d_result;
   
/*   result[0]=1;
   result[1]=1;
   result[2]=1;*/
   
   size_t size  =sizeof(double);
   //allocate memory for arrays on device 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_X,        size*HIT)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Y,        size*HIT)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Z,        size*HIT)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Zerr,     size*HIT)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Mx,       size)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_My,       size)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_M0,       size)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_result,   size*8)); 
  


   CUDA_SAFE_CALL( hipMemcpy(d_X,          X,   size*HIT, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Y,          Y,   size*HIT, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Z,          Z,   size*HIT, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Zerr,       Zerr,size*HIT, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Mx,         Mx,  size,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_My,         My,  size,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_M0,         M0,  size,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_result, result,  size*8,    hipMemcpyHostToDevice));  
  

   dim3 dimBlock2(HIT, 1);
   dim3 dimGrid2(1,1);

   Fit<<< dimGrid2, dimBlock2 >>> (d_X, d_Y,d_Z, d_Zerr,d_Mx,d_My,d_M0, d_result);


   CUDA_SAFE_CALL(hipMemcpy(result, d_result, size*8, hipMemcpyDeviceToHost) );
 
 //  printf(" %f      %f      %f  %   f      %f     %f     %f       %f \n", result[0], result[1],result[2], result[3] ,result[4],result[5] ,result[6],result[7]);

   CUDA_SAFE_CALL( hipFree(d_X)); 
   CUDA_SAFE_CALL( hipFree(d_Y)); 
   CUDA_SAFE_CALL( hipFree(d_Z)); 
   CUDA_SAFE_CALL( hipFree(d_Zerr)); 
   CUDA_SAFE_CALL( hipFree(d_Mx)); 
   CUDA_SAFE_CALL( hipFree(d_My)); 
   CUDA_SAFE_CALL( hipFree(d_M0)); 
   CUDA_SAFE_CALL( hipFree(d_result)); 

}


extern "C" void CircleFitGAllD(double X[TRK*HIT], double Y[TRK*HIT], double Z[TRK*HIT], double Zerr[TRK*HIT], double Mx[TRK],double My[TRK],double M0[TRK],double result[TRK*8])
{
//   printf(" Now in Cuda :  Mx =  %g  :  My =  %g  \n", Mx[0],My[0] );
/* for(int j=0; j<100; j++){
   for(int i=0; i<HIT; i++){
       printf("%d  Zerr[%i] =  %g  :  Z[%i] =  %g  \n", j ,i, Zerr[i+HIT*j],i ,Z[i+HIT*j]  );
   }
 }
*/
   double *d_X;
   double *d_Y;
   double *d_Z;
   double *d_Zerr;
   double *d_Mx;
   double *d_My;
   double *d_M0;
   double *d_result;
   
/*   result[0]=1;
   result[1]=1;
   result[2]=1;*/
   
   size_t size  =sizeof(double);
   //allocate memory for arrays on device 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_X,        size*HIT*TRK)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Y,        size*HIT*TRK)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Z,        size*HIT*TRK)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Zerr,     size*HIT*TRK)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Mx,       size*TRK)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_My,       size*TRK)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_M0,       size*TRK)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_result,   size*8*TRK)); 
  


   CUDA_SAFE_CALL( hipMemcpy(d_X,          X,   size*HIT*TRK, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Y,          Y,   size*HIT*TRK, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Z,          Z,   size*HIT*TRK, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Zerr,       Zerr,size*HIT*TRK, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Mx,         Mx,  size*TRK,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_My,         My,  size*TRK,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_M0,         M0,  size*TRK,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_result, result,  size*8*TRK,    hipMemcpyHostToDevice));  
  
   
   int threads=HIT;
   int tracks=TRK;
   dim3 dimBlock2(threads, 1);
   dim3 dimGrid2(tracks,1);
 
  // FitAllD<<< dimGrid2, dimBlock2 >>> (d_X, d_Y,d_Z, d_Zerr,d_Mx,d_My,d_M0, d_result);
   
  // hipDeviceSynchronize();
   printf(" Now calling the device  \n");
   CUDA_SAFE_CALL(hipMemcpy(result, d_result, size*8*TRK, hipMemcpyDeviceToHost) );
 
 /*  for(int j=0; j<tracks; j++){
     printf("%d   ",j);
     for(int i=0; i<8; i++){
        printf("  %f ", result[i+8*j]);
     }
     printf(" \n");
   }
*/
//   printf(" Now cleaning device memory  \n");
   CUDA_SAFE_CALL( hipFree(d_X)); 
   CUDA_SAFE_CALL( hipFree(d_Y)); 
   CUDA_SAFE_CALL( hipFree(d_Z)); 
   CUDA_SAFE_CALL( hipFree(d_Zerr)); 
   CUDA_SAFE_CALL( hipFree(d_Mx)); 
   CUDA_SAFE_CALL( hipFree(d_My)); 
   CUDA_SAFE_CALL( hipFree(d_M0)); 
   CUDA_SAFE_CALL( hipFree(d_result)); 
//   printf(" Finish cleaning device memory  \n");
}



extern "C" void CircleFitGAllF(float X[TRK*HIT], float Y[TRK*HIT], float Z[TRK*HIT], float Zerr[TRK*HIT], float Mx[TRK],float My[TRK],float M0[TRK],float result[8*TRK])
{
//   printf(" Now in Cuda :  Mx =  %g  :  My =  %g  \n", Mx[0],My[0] );
/* for(int j=0; j<100; j++){
   for(int i=0; i<HIT; i++){
       printf("%d  Zerr[%i] =  %g  :  Z[%i] =  %g  \n", j ,i, Zerr[i+HIT*j],i ,Z[i+HIT*j]  );
   }
 }
*/
   float *d_X;
   float *d_Y;
   float *d_Z;
   float *d_Zerr;
   float *d_Mx;
   float *d_My;
   float *d_M0;
   float *d_result;
   
/*   result[0]=1;
   result[1]=1;
   result[2]=1;*/
   
   size_t size  =sizeof(float);
   //allocate memory for arrays on device 
   hipMalloc((void **) &d_X,        size*HIT*TRK); 
   hipMalloc((void **) &d_Y,        size*HIT*TRK); 
   hipMalloc((void **) &d_Z,        size*HIT*TRK); 
   hipMalloc((void **) &d_Zerr,     size*HIT*TRK); 
   hipMalloc((void **) &d_Mx,       size*TRK); 
   hipMalloc((void **) &d_My,       size*TRK); 
   hipMalloc((void **) &d_M0,       size*TRK); 
   hipMalloc((void **) &d_result,   size*8*TRK); 
  


   hipMemcpy(d_X,          X,   size*HIT*TRK, hipMemcpyHostToDevice);  
   hipMemcpy(d_Y,          Y,   size*HIT*TRK, hipMemcpyHostToDevice);  
   hipMemcpy(d_Z,          Z,   size*HIT*TRK, hipMemcpyHostToDevice);  
   hipMemcpy(d_Zerr,       Zerr,size*HIT*TRK, hipMemcpyHostToDevice);  
   hipMemcpy(d_Mx,         Mx,  size*TRK,    hipMemcpyHostToDevice);  
   hipMemcpy(d_My,         My,  size*TRK,    hipMemcpyHostToDevice);  
   hipMemcpy(d_M0,         M0,  size*TRK,    hipMemcpyHostToDevice);  
   hipMemcpy(d_result, result,  size*8*TRK,  hipMemcpyHostToDevice);  
  
   
   int threads=HIT;
   int tracks=TRK;
   dim3 dimBlock2(threads, 1);
   dim3 dimGrid2(tracks,1);

  // FitAllF<<< dimGrid2, dimBlock2 >>> (d_X, d_Y,d_Z, d_Zerr,d_Mx,d_My,d_M0, d_result);
   
//   hipDeviceSynchronize();

   hipMemcpy(result, d_result, size*8*TRK, hipMemcpyDeviceToHost);
 
 /*  for(int j=0; j<tracks; j++){
     printf("%d   ",j);
     for(int i=0; i<8; i++){
        printf("  %f ", result[i+8*j]);
     }
     printf(" \n");
   }
*/
  // printf(" Now cleaning device memory  \n");
   CUDA_SAFE_CALL( hipFree(d_X)); 
   CUDA_SAFE_CALL( hipFree(d_Y)); 
   CUDA_SAFE_CALL( hipFree(d_Z)); 
   CUDA_SAFE_CALL( hipFree(d_Zerr)); 
   CUDA_SAFE_CALL( hipFree(d_Mx)); 
   CUDA_SAFE_CALL( hipFree(d_My)); 
   CUDA_SAFE_CALL( hipFree(d_M0)); 
   CUDA_SAFE_CALL( hipFree(d_result)); 
 //   printf(" Finish cleaning device memory  \n");
}




extern "C" void CircleFitGF(float X[HIT], float Y[HIT], float Z[HIT], float Zerr[HIT], float *Mx,float *My,float *M0,float *result)
{
//   printf(" Now in Cuda :  Mx =  %g  :  My =  %g  \n", Mx[0],My[0] );
 
 /*  for(int i=0; i<50; i++){
       printf(" Zerr[%i] =  %g  :  Z[%i] =  %g  \n", i, Zerr[i],i ,Z[i]  );
   }
*/
   float *d_X;
   float *d_Y;
   float *d_Z;
   float *d_Zerr;
   float *d_Mx;
   float *d_My;
   float *d_M0;
   float *d_result;
   
/*   result[0]=1;
   result[1]=1;
   result[2]=1;*/
   
   size_t size  =sizeof(float);
   //allocate memory for arrays on device 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_X,        size*HIT)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Y,        size*HIT)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Z,        size*HIT)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Zerr,     size*HIT)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_Mx,       size)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_My,       size)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_M0,       size)); 
   CUDA_SAFE_CALL( hipMalloc((void **) &d_result,   size*8)); 
  


   CUDA_SAFE_CALL( hipMemcpy(d_X,          X,   size*HIT, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Y,          Y,   size*HIT, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Z,          Z,   size*HIT, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Zerr,       Zerr,size*HIT, hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_Mx,         Mx,  size,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_My,         My,  size,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_M0,         M0,  size,    hipMemcpyHostToDevice));  
   CUDA_SAFE_CALL( hipMemcpy(d_result, result,  size*8,    hipMemcpyHostToDevice));  
  

   dim3 dimBlock2(HIT, 1);
   dim3 dimGrid2(1,1);

   FitF<<< dimGrid2, dimBlock2 >>> (d_X, d_Y,d_Z, d_Zerr,d_Mx,d_My,d_M0, d_result);


   CUDA_SAFE_CALL(hipMemcpy(result, d_result, size*8, hipMemcpyDeviceToHost) );
 
 //  printf(" %f      %f      %f  %   f      %f     %f     %f       %f \n", result[0], result[1],result[2], result[3] ,result[4],result[5] ,result[6],result[7]);

   CUDA_SAFE_CALL( hipFree(d_X)); 
   CUDA_SAFE_CALL( hipFree(d_Y)); 
   CUDA_SAFE_CALL( hipFree(d_Z)); 
   CUDA_SAFE_CALL( hipFree(d_Zerr)); 
   CUDA_SAFE_CALL( hipFree(d_Mx)); 
   CUDA_SAFE_CALL( hipFree(d_My)); 
   CUDA_SAFE_CALL( hipFree(d_M0)); 
   CUDA_SAFE_CALL( hipFree(d_result)); 

}
















